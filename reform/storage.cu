#include "hip/hip_runtime.h"
﻿#include <storage.cuh>
#include <utils.cuh>

#include <hiprand/hiprand_kernel.h>
#include <>

#include <cmath>

Storage::Storage(const std::vector<int> &_shape) : shape(_shape) {
  int size = 1;
  for (int i = 0; i < _shape.size(); i++) {
    size *= _shape[i];
  }

  this->data.resize(size);
}

Storage::Storage(const std::vector<int> &_shape, float value) : shape(_shape) {
  int size = 1;
  for (int i = 0; i < _shape.size(); i++) {
    size *= _shape[i];
  }

  this->data.resize(size, value);
}

Storage::Storage(const std::vector<int> &_shape,
                 const std::vector<float> &_data)
    : shape(_shape), data(_data.begin(), _data.end()) {
  this->check_size();
}

Storage::Storage(const Storage &other) { *this = other; }

Storage &Storage::operator=(const Storage &other) {
  if (this != &other) {
    this->shape = other.shape;
    this->data = other.data;
  }

  return *this;
}

Storage::Storage(Storage &&other) { *this = std::move(other); }

Storage &Storage::operator=(Storage &&other) {
  if (this != &other) {
    this->shape = std::move(other.shape);
    this->data = std::move(other.data);
  }
  return *this;
}

void Storage::reshape(const std::vector<int> &_shape) {
  this->shape = _shape;
  this->check_size();
}

void Storage::resize(const std::vector<int> &_shape) {
  this->shape = _shape;

  int size = 1;
  for (int i = 0; i < _shape.size(); i++) {
    size *= _shape[i];
  }

  if (size != this->data.size()) {
    this->data.resize(size);
  }
}

__global__ void storage_xavier(float *a, int size, float scale,
                               hiprandState *cs) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    hiprand_init(1234, index, 0, &cs[index]);
    a[index] = (hiprand_uniform(&cs[index]) * 2 - 1) * scale;
  }
}

void Storage::xavier(size_t in_size, size_t out_size) {
  float *a_ptr = RAW_PTR(this->data);
  int size = this->data.size();
  int grid_size = ceil((float)(size) / BLOCK_SIZE);

  thrust::device_vector<hiprandState> cs(size);
  hiprandState *cs_ptr = RAW_PTR(cs);
  float scale = std::sqrt((float)6) / std::sqrt((float)(in_size) + out_size);
  storage_xavier<<<grid_size, BLOCK_SIZE>>>(a_ptr, size, scale, cs_ptr);

  CUDA_POST_KERNEL_CHECK;
}

void Storage::check_size() {
  int size = 1;
  for (int i = 0; i < this->shape.size(); i++) {
    size *= this->shape[i];
  }
  CHECK_EQ(size, this->data.size(), "Storage: size error");
}