#include "hip/hip_runtime.h"
                                                                                                                                    #include <linear.cuh>
#include <memory>

void operator_linear(const Storage *inputs, const Storage *weights,
                     Storage *output) {
  operator_matmul(inputs, weights, output);
}

void operator_d_linear(
    const Storage *outputs_grad, const Storage *inputs, const Storage *weights,
    Storage *weights_grad, Storage *inputs_grad,
    std::unordered_map<std::string, std::unique_ptr<Storage>> &temp) {
  // W^T
  std::vector<int> weights_t_shape{weights->get_shape()[1],
                                   weights->get_shape()[0]};
  INIT_TEMP(temp, "weights_t", weights_t_shape);
  operator_transpose(weights, temp["weights_t"].get());

  // X^T
  std::vector<int> inputs_t_shape(
      {inputs->get_shape()[1], inputs->get_shape()[0]});
  INIT_TEMP(temp, "inputs_t", inputs_t_shape);
  operator_transpose(inputs, temp["inputs_t"].get());

  // Y = X * W
  // dL/dX = dL/dY * W^T
  // dL/dW = X^T * dL/dY
  operator_matmul(outputs_grad, temp["weights_t"].get(), inputs_grad);
  operator_matmul(temp["inputs_t"].get(), outputs_grad, weights_grad);
}

__global__ void operator_bias_h(const float *inputs, const float *bias,
                                float *output, int width, int size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < size) {
    int col = index % width;
    output[index] = inputs[index] + bias[col];
  }
}

void operator_linear_bias(const Storage *inputs, const Storage *bias,
                          Storage *output) {
  const float *inputs_ptr = RAW_PTR(inputs->get_data());
  const float *bias_ptr = RAW_PTR(bias->get_data());
  float *output_ptr = RAW_PTR(output->get_data());

  int size = inputs->get_data().size();
  int grid_size = ceil((float)(size) / BLOCK_SIZE);
  int width = bias->get_data().size();
  operator_bias_h<<<grid_size, BLOCK_SIZE>>>(inputs_ptr, bias_ptr, output_ptr,
                                             width, size);

  CUDA_POST_KERNEL_CHECK;
}

void operator_d_linear_bias(const Storage *outputs_grad, Storage *bias_grad) {
  operator_sum(outputs_grad, 0, bias_grad);
}

Linear::Linear(int in_size, int out_size, bool is_bias)
    : in_size(in_size), out_size(out_size), is_bias(is_bias) {
  this->weights.reset(new Storage({in_size, out_size}));
  this->weights_grad.reset(new Storage({in_size, out_size}));
  this->weights->xavier(in_size, out_size);

  if (this->is_bias) {
    this->bias.reset(new Storage({1, out_size}));
    this->bias_grad.reset(new Storage({1, out_size}));
    this->bias->xavier(in_size, out_size);
  }
}

std::vector<std::pair<Storage *, Storage *>> Linear::parameters() {
  if (this->is_bias) {
    return {std::make_pair(this->weights.get(), this->weights_grad.get()),
            std::make_pair(this->bias.get(), this->bias_grad.get())};
  } else {
    return {std::make_pair(this->weights.get(), this->weights_grad.get())};
  }
}

void Linear::forward() {
  const Storage *input = this->pre->get_output();
  std::vector<int> output_shape = {input->get_shape()[0], this->out_size};

  INIT_STORAGE(this->output, output_shape);

  operator_linear(input, this->weights.get(), this->output.get());
  if (this->bias) {
    operator_linear_bias(this->output.get(), this->bias.get(),
                         this->output.get());
  }
}

void Linear::backward() {
  const Storage *input = this->pre->get_output();
  const Storage *output_grad = this->next->get_grad();

  INIT_STORAGE(this->grad, input->get_shape());

  if (this->bias) {
    operator_d_linear_bias(output_grad, this->bias_grad.get());
  }

  operator_d_linear(output_grad, input, this->weights.get(),
                    this->weights_grad.get(), this->grad.get(), this->temp);
}